#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:17:12 on Sun, Nov 12, 2023
//
// Description: decoding fp8 fwd hdim256

#include "decoding_attn_fp8/decoding_fp8_fwd_launch_template.h"

#ifdef FAI_ENABLE_FP8

template <>
void run_mha_decoding_fp8_fwd_<256, __hip_fp8_e5m2_fnuz>(const DecodingFP8Params<__hip_fp8_e5m2_fnuz> &params) {
    mha_decoding_fp8_fwd<256, 256, 16, __hip_fp8_e5m2_fnuz>(params);
}

template <>
void run_mha_decoding_fp8_fwd_<256, __hip_fp8_e4m3_fnuz>(const DecodingFP8Params<__hip_fp8_e4m3_fnuz> &params) {
    mha_decoding_fp8_fwd<256, 256, 16, __hip_fp8_e4m3_fnuz>(params);
}

#endif
