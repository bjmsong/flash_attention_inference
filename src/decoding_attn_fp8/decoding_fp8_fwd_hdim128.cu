#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:17:12 on Sun, Nov 12, 2023
//
// Description: decoding fp8 fwd hdim128

#include "decoding_attn_fp8/decoding_fp8_fwd_launch_template.h"

#ifdef FAI_ENABLE_FP8

template <>
void run_mha_decoding_fp8_fwd_<128, __hip_fp8_e5m2_fnuz>(const DecodingFP8Params<__hip_fp8_e5m2_fnuz> &params) {
    if (params.b <= 4) {
        mha_decoding_fp8_fwd<128, 256, 8, __hip_fp8_e5m2_fnuz>(params);
    } else {
        mha_decoding_fp8_fwd<128, 128, 16, __hip_fp8_e5m2_fnuz>(params);
    }
}

template <>
void run_mha_decoding_fp8_fwd_<128, __hip_fp8_e4m3_fnuz>(const DecodingFP8Params<__hip_fp8_e4m3_fnuz> &params) {
    if (params.b <= 4) {
        mha_decoding_fp8_fwd<128, 256, 8, __hip_fp8_e4m3_fnuz>(params);
    } else {
        mha_decoding_fp8_fwd<128, 128, 16, __hip_fp8_e4m3_fnuz>(params);
    }
}

#endif
