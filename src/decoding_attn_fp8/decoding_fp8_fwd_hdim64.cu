#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:17:12 on Sun, Nov 12, 2023
//
// Description: decoding fp8 fwd hdim64

#include "decoding_attn_fp8/decoding_fp8_fwd_launch_template.h"

#ifdef FAI_ENABLE_FP8

template <>
void run_mha_decoding_fp8_fwd_<64, __hip_fp8_e5m2_fnuz>(const DecodingFP8Params<__hip_fp8_e5m2_fnuz> &params) {
    mha_decoding_fp8_fwd<64, 256, 4, __hip_fp8_e5m2_fnuz>(params);
}

template <>
void run_mha_decoding_fp8_fwd_<64, __hip_fp8_e4m3_fnuz>(const DecodingFP8Params<__hip_fp8_e4m3_fnuz> &params) {
    mha_decoding_fp8_fwd<64, 256, 4, __hip_fp8_e4m3_fnuz>(params);
}

#endif
